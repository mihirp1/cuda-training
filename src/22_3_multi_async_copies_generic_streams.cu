//Author: Ugo Varetto
//Launch multiple asynchronous memcopies on different gpus
//and execute kernel. 
//Specify total buffer size in bytes and list of gpu ids
//on the command line.
//NOTE: the number of gpu threads used is always 1024 so the
//per-gpu buffer size (=total size / num gpus) *must* be
//evenly divisible by 1024.
//@todo automatically compute a valid thread count from
//buffer size
//
//Verify (with nvvp) that transfers happen in parallel


#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <vector>
#include <algorithm>
#include <cstdlib>

using namespace std;

typedef signed char Int8;

#ifndef PEER_ACCESS
__global__
void Negate(Int8* buffer) {
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    buffer[i] = -buffer[i];
}
#else
__global__
void Negate(Int8* src, Int8* dest) {
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    dest[i] = -src[i];
}
#endif

void InitHostBuffer(Int8* buf, size_t hostSize, int numDevices) {
    const size_t devSize = hostSize / numDevices;
    assert(devSize);
    for(int i = 0; i != numDevices; ++i) {
        fill(buf + i * devSize, buf + i * devSize + devSize, Int8(-(i+1)));
    }
}

void EnablePeerAccess(const vector< int >& devices, int src) {
    assert(!devices.empty());
    assert(src < int(devices.size()));
    assert(src >= 0);
    int curDevice = -1;
    assert(hipGetDevice(&curDevice) == hipSuccess);
    assert(hipSetDevice(devices[src]) == hipSuccess);
    for(int i = 0; i != devices.size(); ++i) {
        const int PEER_DEVICE_TO_ACCESS = devices[i];
        const int PEER_ACCESS_FLAGS = 0;
        assert(hipDeviceEnablePeerAccess(PEER_DEVICE_TO_ACCESS, PEER_ACCESS_FLAGS)
               == hipSuccess); 
         
    }
    assert(hipSetDevice(curDevice) == hipSuccess);
}



void EnableAllToAllPeerAccess(const vector< int >& devices) {
    for(int i = 0; i != devices.size(); ++i) EnablePeerAccess(devices, i);
}


int main(int argc, char** argv) {
    assert(sizeof(Int8) == 1);
    if(argc < 2) {
        cout << "usage: " << argv[0] << " <total buffer size> <gpu ids>" << endl;
        exit(EXIT_FAILURE);
    }
    vector< int > gpus(argc - 2, -1);
    for(int i = 2; i != argc; ++i) {
        gpus[i - 2] = atoi(argv[i]);
    }
    const size_t requestedBufferSize = atoll(argv[1]);
    const int requestedNumDevices = gpus.size();
    //allocate pinned host buffer
    const size_t HOST_BUFFER_SIZE = requestedBufferSize < 1 ? 
                                    size_t(1) << 32 : requestedBufferSize;
    const int NUM_DEVICES = requestedNumDevices < 1 ? 4 : requestedNumDevices;
    const size_t DEVICE_BUFFER_SIZE = HOST_BUFFER_SIZE / NUM_DEVICES;
    assert(DEVICE_BUFFER_SIZE);
    cout << "Number of devices:      " << NUM_DEVICES << endl
         << "Buffer size:            " << HOST_BUFFER_SIZE << endl
         << "Per-device buffer size: " << DEVICE_BUFFER_SIZE << endl;
    if(HOST_BUFFER_SIZE % NUM_DEVICES != 0) {
        cout << "WARNING: buffer size NOT "
                "evenly divisible by device buffer size" << endl;
    }
    Int8* hostBuffer = 0;
    hipError_t err = hipHostMalloc((void**) &hostBuffer, HOST_BUFFER_SIZE, hipHostMallocDefault);
    assert(hostBuffer);
    assert(err == hipSuccess);
    //initialize host buffer with -1-1-1-1-2-2-2-2-3-3-3-3-4-4-4-4
    InitHostBuffer(hostBuffer, HOST_BUFFER_SIZE, NUM_DEVICES);
    //allocate 4 device buffers, one per device
    vector< Int8* > deviceBuffers(NUM_DEVICES, (Int8*)(0));
    vector< hipStream_t > streams(NUM_DEVICES, hipStream_t());
    for(int d = 0; d != NUM_DEVICES; ++d) {
        const int gpu = gpus[d];
        err = hipSetDevice(gpu);
        assert(err == hipSuccess);
        err = hipMalloc((void**) &deviceBuffers[d], DEVICE_BUFFER_SIZE);
        assert(deviceBuffers[d]);
        assert(err == hipSuccess);
        err = hipStreamCreate(&streams[d]);
        assert(err == hipSuccess);
    }
    //optioanlly enable peer access
#ifdef PEER_ACCESS
    EnableAllToAllPeerAccess(gpus.begin());
#endif     
    //async per-device copies
    for(int d = 0; d != NUM_DEVICES; ++d) {
        const int gpu = gpus[d];
        err = hipSetDevice(gpu);
        assert(err == hipSuccess);
        err = hipMemcpyAsync(deviceBuffers[d], 
                              hostBuffer + d * DEVICE_BUFFER_SIZE,
                              DEVICE_BUFFER_SIZE, 
                              hipMemcpyHostToDevice,
                              streams[d]);
        assert(err == hipSuccess);
    }
#ifdef PEER_ACCESS    
    cudaDeviceSynchronize();
#endif
    const int THREAD_BLOCK_SIZE = 1024;
    const int BLOCK_SIZE = DEVICE_BUFFER_SIZE / THREAD_BLOCK_SIZE;
    for(int d = 0; d != NUM_DEVICES; ++d) {
        const int gpu = gpus[d];
        err = hipSetDevice(gpu);
        assert(err == hipSuccess);
        const bool KERNEL_ENABLED_OPTION = d == 0; //only enable for first device
#ifdef PEER_ACCESS
        Negate<<< BLOCK_SIZE, THREAD_BLOCK_SIZE, 0, streams[d] >>>(
            deviceBuffers[(d + 1) % NUM_DEVICES], deviceBuffers[d]);
#else
        Negate<<< BLOCK_SIZE, THREAD_BLOCK_SIZE, 0, streams[d] >>>(deviceBuffers[d]);
#endif        
#ifdef CHECK_KERNEL_LAUNCH       
        err == cudaGetLastError(); //no idea about what this does, does it trigger a barrier ?
        assert(err == cudaSuccess);
#endif
    }
    //
    for(int d = 0; d != NUM_DEVICES; ++d) {
        const int gpu = gpus[d];
        err = hipSetDevice(gpu);
        assert(err == hipSuccess);
        err = hipMemcpyAsync(hostBuffer + d * DEVICE_BUFFER_SIZE,
                              deviceBuffers[d], 
                              DEVICE_BUFFER_SIZE,
                              hipMemcpyDeviceToHost,
                              streams[d]);
        assert(err == hipSuccess);
    }

    for(int d = 0; d != NUM_DEVICES; ++d) {
        const int gpu = gpus[d];
        err = hipSetDevice(gpu);
        assert(err == hipSuccess);
        err = hipDeviceSynchronize();
        assert(err == hipSuccess);
    }
#ifdef PEER_ACCESS
    for(int d = 0; d != NUM_DEVICES; ++d) {
        for(Int8* p = hostBuffer + d * DEVICE_BUFFER_SIZE;
            p != hostBuffer + d * DEVICE_BUFFER_SIZE + DEVICE_BUFFER_SIZE;
            ++p);// assert(*p == 1);
    }
#else
    for(int d = 0; d != NUM_DEVICES; ++d) {
        for(Int8* p = hostBuffer + d * DEVICE_BUFFER_SIZE;
            p != hostBuffer + d * DEVICE_BUFFER_SIZE + DEVICE_BUFFER_SIZE;
            ++p) assert(*p == (d + 1));
    }
#endif    
    err = hipHostFree(hostBuffer);
    assert(err == hipSuccess);
    for(int d = 0; d != NUM_DEVICES; ++d) {
        const int gpu = gpus[d];
        err = hipSetDevice(gpu);
        assert(err == hipSuccess);
        err = hipFree(deviceBuffers[d]);
        err = hipStreamDestroy(streams[d]);
        assert(err == hipSuccess);
    }
    err = hipDeviceReset();
    assert(err == hipSuccess);
    cout << "PASSED" << endl;
    return 0;
}
