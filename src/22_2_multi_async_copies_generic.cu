//Author: Ugo Varetto
//Launch multiple asynchronous memcopies on four different gpus
//and execute kernel.
//Verify (with nvvp) that transfers happen in parallel


#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <vector>
#include <algorithm>
#include <cstdlib>
#include <map>

using namespace std;

typedef signed char Int8;

__global__
void Negate(Int8* buffer) {
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    buffer[i] = -buffer[i];
}


void InitHostBuffer(Int8* buf, size_t hostSize, int numDevices) {
    const size_t devSize = hostSize / numDevices;
    assert(devSize);
    for(int i = 0; i != numDevices; ++i) {
        fill(buf + i * devSize, buf + i * devSize + devSize, Int8(-(i+1)));
    }
}


int main(int argc, char** argv) {
    assert(sizeof(Int8) == 1);
    if(argc < 2) {
        cout << "usage: " << argv[0] << " <total buffer size> <gpu ids>" << endl;
        exit(EXIT_FAILURE);
    }
    map< int, int > gpus;
    for(int i = 2; i != argc; ++i) {
        gpus[i - 2] = atoi(argv[i]);
    }
    const size_t requestedBufferSize = atoll(argv[1]);
    const int requestedNumDevices = gpus.size();
    //allocate pinned host buffer
    const size_t HOST_BUFFER_SIZE = requestedBufferSize < 1 ? 
                                    size_t(1) << 32 : requestedBufferSize;
    const int NUM_DEVICES = requestedNumDevices < 1 ? 4 : requestedNumDevices;
    const size_t DEVICE_BUFFER_SIZE = HOST_BUFFER_SIZE / NUM_DEVICES;
    assert(DEVICE_BUFFER_SIZE);
    cout << "Number of devices:      " << NUM_DEVICES << endl
         << "Buffer size:            " << HOST_BUFFER_SIZE << endl
         << "Per-device buffer size: " << DEVICE_BUFFER_SIZE << endl;
    if(HOST_BUFFER_SIZE % NUM_DEVICES != 0) {
        cout << "WARNING: buffer size NOT "
                "evenly divisible by device buffer size" << endl;
    }
    Int8* hostBuffer = 0;
    hipError_t err = hipHostMalloc((void**) &hostBuffer, HOST_BUFFER_SIZE, hipHostMallocDefault);
    assert(hostBuffer);
    assert(err == hipSuccess);
    //initialize host buffer with -1-1-1-1-2-2-2-2-3-3-3-3-4-4-4-4
    InitHostBuffer(hostBuffer, HOST_BUFFER_SIZE, NUM_DEVICES);
    //allocate 4 device buffers, one per device
    vector< Int8* > deviceBuffers(NUM_DEVICES, (Int8*)(0));
    for(int d = 0; d != NUM_DEVICES; ++d) {
        const int gpu = gpus[d];
        err = hipSetDevice(gpu);
        assert(err == hipSuccess);
        err = hipMalloc((void**) &deviceBuffers[d], DEVICE_BUFFER_SIZE);
        assert(deviceBuffers[d]);
        assert(err == hipSuccess);
    }
    //async per-device copies
    for(int d = 0; d != NUM_DEVICES; ++d) {
        const int gpu = gpus[d];
        err = hipSetDevice(gpu);
        assert(err == hipSuccess);
        err = hipMemcpyAsync(deviceBuffers[d], 
                              hostBuffer + d * DEVICE_BUFFER_SIZE,
                              DEVICE_BUFFER_SIZE, hipMemcpyHostToDevice);
        assert(err == hipSuccess);
    }
    //
    const int THREAD_BLOCK_SIZE = 1024;
    const int BLOCK_SIZE = DEVICE_BUFFER_SIZE / THREAD_BLOCK_SIZE;
    for(int d = 0; d != NUM_DEVICES; ++d) {
        const int gpu = gpus[d];
        err = hipSetDevice(gpu);
        assert(err == hipSuccess);
        Negate<<< BLOCK_SIZE, THREAD_BLOCK_SIZE >>>(deviceBuffers[d]);
#ifdef CHECK_KERNEL_LAUNCH       
        err == cudaGetLastError(); //no idea about what this does, does it trigger a barrier ?
        assert(err == cudaSuccess);
#endif
    }
    //
    for(int d = 0; d != NUM_DEVICES; ++d) {
        const int gpu = gpus[d];
        err = hipSetDevice(gpu);
        assert(err == hipSuccess);
        err = hipMemcpyAsync(hostBuffer + d * DEVICE_BUFFER_SIZE,
                              deviceBuffers[d], 
                              DEVICE_BUFFER_SIZE, hipMemcpyDeviceToHost);
        assert(err == hipSuccess);
    }

    for(int d = 0; d != NUM_DEVICES; ++d) {
        const int gpu = gpus[d];
        err = hipSetDevice(gpu);
        assert(err == hipSuccess);
        err = hipDeviceSynchronize();
        assert(err == hipSuccess);
    }
    
    for(int d = 0; d != NUM_DEVICES; ++d) {
        for(Int8* p = hostBuffer + d * DEVICE_BUFFER_SIZE;
            p != hostBuffer + d * DEVICE_BUFFER_SIZE + DEVICE_BUFFER_SIZE;
            ++p) assert(*p == (d + 1));
    }

    err = hipHostFree(hostBuffer);
    assert(err == hipSuccess);
    for(int d = 0; d != NUM_DEVICES; ++d) {
        const int gpu = gpus[d];
        err = hipSetDevice(gpu);
        assert(err == hipSuccess);
        err = hipFree(deviceBuffers[d]);
        assert(err == hipSuccess);
    }
    err = hipDeviceReset();
    assert(err == hipSuccess);
    cout << "PASSED" << endl;
    return 0;
}
